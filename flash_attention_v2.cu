#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__
void flash_attention_2_forward_kernel(
    const float* Q,
    const float* K,
    const float* V,
    const int N,
    const int d,
    const int Tc,
    const int Tr,
    const int Bc,
    const int Br,
    const float softmax_scale,
    float* L,
    float* O
) {
    int tx = threadIdx.x;
    int bx = blockIdx.x; int by = blockIdx.y;  // batch and head index

    // Offset into Q,K,V,O - different for each batch and head
    int qkv_offset = (bx * gridDim.y * N * d) + (by * N * d);  // gridDim.y = nh
    int lm_offset = (bx * gridDim.y * N) + (by * N);  // offset for L

    // Define SRAM for Q,K,V,S
    extern __shared__ float sram[];
    int tile_size = Bc * d;  // size of Qi, Kj, Vj
    float* Qi = sram;
    float* Kj = &sram[tile_size];
    float* Vj = &sram[tile_size * 2];
    float* S = &sram[tile_size * 3];

    for (int i = 0; i < Tr; ++i) {
        if (i * Br + tx >= N)
            break;  // break if we are done with the sequence

        // Load Qi from HBM to SRAM, l and m to registers
        for (int x = 0; x < d; x++) {
            Qi[(tx * d) + x] = Q[qkv_offset + (tile_size * i) + (tx * d) + x];
        }
        float row_m_prev = -INFINITY;
        float row_l_prev = 0;

        // Causal mask: j <= i
        for (int j = 0; j <= i; ++j) {
            __syncthreads();
            // Load Kj, Vj from HBM to SRAM
            for (int x = 0; x < d; x++) {
                Kj[(tx * d) + x] = K[qkv_offset + (tile_size * j) + (tx * d) + x];
                Vj[(tx * d) + x] = V[qkv_offset + (tile_size * j) + (tx * d) + x];
            }
            __syncthreads();
            // S_i^j = softmax_scale * QiKj^T
            // S_i^j[tx][y] = softmax_scale * Sum_{x = 0}^{d-1} Qi[tx][x] * Kj[y][x]
            float row_m = -INFINITY;
            for (int y = 0; y < Bc; y++) {
                if (j * Bc + y >= N)
                    break;  // break if we are done with the sequence
                if (i * Br + tx < j * Bc + y)
                    break;
                float sum = 0;
                for (int x = 0; x < d; x++)
                    sum += Qi[(tx * d) + x] * Kj[(y * d) + x];
                sum *= softmax_scale;
                S[(Bc * tx) + y] = sum;

                if (sum > row_m)
                    row_m = sum;
            }

            // m_i^j = max(m_i^j-1, row_max(S_i^j))
            float new_row_m = max(row_m_prev, row_m);

            // P_i^j = exp(S_i^j - m_i^j)
            // P_i^j[tx][y] = exp(S_i^j[tx][y] - m_i^j)
            float row_l = 0;
            for (int y = 0; y < Bc; y++) {
                if (j * Bc + y >= N)
                    break;  // break if we are done with the sequence
                if (i * Br + tx < j * Bc + y)
                    break;
                S[(Bc * tx) + y] = __expf(S[(Bc * tx) + y] - new_row_m);
                row_l += S[(Bc * tx) + y];
            }

            // l_i^j = (exp(m_i^j-1 - m_i^j) * l_i^j-1) + row_sum(P_i^j)
            float row_m_exp = __expf(row_m_prev - new_row_m);
            float new_row_l = (row_m_exp * row_l_prev) + row_l;

            // O_i^j = diag(exp(m_i^j-1 - m_i^j))^-1 * O_i^j-1 + P_i^jVj
            for (int x = 0; x < d; x++) {
                float pv = 0;  // Pij * Vj
                for (int y = 0; y < Bc; y++) {
                    if (j * Bc + y >= N)
                        break;  // break if we are done with the sequence
                    if (i * Br + tx < j * Bc + y)
                        break;
                    pv += S[(Bc * tx) + y] * Vj[(y * d) + x];
                }
                O[qkv_offset + (tile_size * i) + (tx * d) + x] = \
                    row_m_exp * O[qkv_offset + (tile_size * i) + (tx * d) + x] + pv;
            }

            // Update m and l
            row_m_prev = new_row_m;
            row_l_prev = new_row_l;
        }

        // O_i = diag(l_i^{Tc})^-1 * O_i^{Tc}
        for (int x = 0; x < d; x++)
            O[qkv_offset + (tile_size * i) + (tx * d) + x] /= row_l_prev;
        // L_i = m_i^{Tc} + log(l_i^{Tc})
        L[lm_offset + (Br * i) + tx] = row_m_prev + __logf(row_l_prev);
    }
}

__global__
void flash_attention_2_backward_kernel(
    const float* Q,
    const float* K,
    const float* V,
    const float* O,
    const float* dO,
    const float* L,
    const int N,
    const int d,
    const int Tc,
    const int Tr,
    const int Bc,
    const int Br,
    const float softmax_scale,
    float* dQ,
    float* dK,
    float* dV
) {
    int tx = threadIdx.x;
    int bx = blockIdx.x; int by = blockIdx.y;  // batch and head index

    // Offset into Q,K,V,O - different for each batch and head
    int qkv_offset = (bx * gridDim.y * N * d) + (by * N * d);  // gridDim.y = nh
    int lm_offset = (bx * gridDim.y * N) + (by * N);  // offset for L

    // Define SRAM for Q,K,V,S
    extern __shared__ float sram[];
    int col_tile_size = Bc * d;  // size of Kj, Vj
    int row_tile_size = Br * d;  // size of Qi
    float* Kj = sram;
    float* Vj = &sram[col_tile_size];

    float* dKj = &sram[col_tile_size * 2];
    float* dVj = &sram[col_tile_size * 3];

    float* Qi = &sram[col_tile_size * 4];
    float* Oi = &sram[col_tile_size * 4 + row_tile_size];
    float* dOi = &sram[col_tile_size * 4 + row_tile_size * 2];

    // We also use S for P. Likewise, we use dS for dP.
    // We can reuse the same memory because we don't need S and P at the same time.
    // We also don't need dS and dP at the same time.
    float* S = &sram[col_tile_size * 4 + row_tile_size * 3];
    float* dS = &sram[col_tile_size * 4 + row_tile_size * 3 + Bc * Br];

    for (int j = 0; j < Tc; j++) {

        // Load Kj, Vj to SRAM
        for (int x = 0; x < d; x++) {
            Kj[(tx * d) + x] = K[qkv_offset + (col_tile_size * j) + (tx * d) + x];
            Vj[(tx * d) + x] = V[qkv_offset + (col_tile_size * j) + (tx * d) + x];
        }

        // Initialize dKj, dVj to 0
        for (int x = 0; x < d; x++) {
            dKj[(tx * d) + x] = 0;
            dVj[(tx * d) + x] = 0;
        }

        for (int i = j; i < Tr; i++)  {
            __syncthreads();
            // Load Qi, Oi, dOi, dQi, li, mi to SRAM
            // Also load l, m to registers
            float Di = 0;
            for (int x = 0; x < d; x++) {
                Qi[(tx * d) + x] = Q[qkv_offset + (row_tile_size * i) + (tx * d) + x];
                Oi[(tx * d) + x] = O[qkv_offset + (row_tile_size * i) + (tx * d) + x];
                dOi[(tx * d) + x] = dO[qkv_offset + (row_tile_size * i) + (tx * d) + x];
                Di += dOi[(tx * d) + x] * Oi[(tx * d) + x];
            }
            float l_curr = L[lm_offset + (Br * i) + tx];

            // Sij = softmax_scale * QiKj^T
            // Sij[tx][y] = softmax_scale * Sum_{y = 0}^{Bc-1} Qi[tx][x] * Kj[y][x]
            for (int y = 0; y < Bc; y++) {
                float sum = 0;
                for (int x = 0; x < d; x++) {
                    sum += Qi[(tx * d) + x] * Kj[(y * d) + x];
                }
                sum *= softmax_scale;
                if (i * Br + tx < j * Bc + y)
                    sum = -INFINITY;
                S[(Bc * tx) + y] = sum;
            }

            // Pij = diag(li)^-1 * exp(Sij - mi)
            // Pij[tx][y] = (1 / li[tx]) * exp(Sij[tx][y] - mi[tx])
            for (int y = 0; y < Bc; y++) {
                if (i * Br + tx < j * Bc + y)
                    S[(Bc * tx) + y] = 0;
                else
                    S[(Bc * tx) + y] = __expf(S[(Bc * tx) + y] - l_curr);
            }
            __syncthreads();
            // dVj <- dVj + Pij^T * dOi
            // dVj[tx][x] = dVj[tx][x] + Sum_{y = 0}^{Br-1} Pij[y][tx] * dOi[tx][x]
            for (int x = 0; x < d; x++) {
                float sum = 0;
                for (int y = 0; y < Br; y++) {
                    sum += S[(Bc * y) + tx] * dOi[(tx * d) + x];
                }
                atomicAdd(&dVj[(tx * d) + x], sum);
            }

            // dPij <- dOi * Vj^T
            // dPij[tx][y] = Sum_{x = 0}^{d-1} dOi[tx][x] * Vj[y][x]
            for (int y = 0; y < Bc; y++) {
                float sum = 0;
                for (int x = 0; x < d; x++) {
                    sum += dOi[(tx * d) + x] * Vj[(y * d) + x];
                }
                dS[(Bc * tx) + y] = sum;
            }

            // dSij <- Pij * (dPij - Di)
            // dSij[tx][y] = Pij[tx][y] * (dPij[tx][y] - Di[tx])
            for (int y = 0; y < Bc; ++y) {
                dS[(Bc * tx) + y] = S[(Bc * tx) + y] * (dS[(Bc * tx) + y] - Di);
            }

            // dQi <- dQi + softmax_scale * dSijKj
            // dQ[tx][x] = dQ[tx][x] + softmax_scale * Sum_{y = 0}^{Bc-1} dSij[tx][y] * Kj[y][x]
            for (int x = 0; x < d; x++) {
                float sum = 0;
                for (int y = 0; y < Bc; y++) {
                    sum += dS[(Bc * tx) + y] * Kj[(y * d) + x];
                }
                sum *= softmax_scale;
                atomicAdd(&dQ[qkv_offset + (row_tile_size * i) + (tx * d) + x], sum);
            }
            __syncthreads();
            // dKj <- dKj + softmax_scale * dSij^TQi
            // dKj[tx][x] = dKj[tx][x] + softmax_scale * Sum_{y = 0}^{Br-1} dSij[y][tx] * Qi[y][x]
            for (int x = 0; x < d; x++) {
                float sum = 0;
                for (int y = 0; y < Br; y++) {
                    sum += dS[(Bc * y) + tx] * Qi[(y * d) + x];
                }
                sum *= softmax_scale;
                atomicAdd(&dKj[(tx * d) + x], sum);
            }
        }

        // Upload Kj, Vj to HRAM
        for (int x = 0; x < d; x++) {
            dK[qkv_offset + (row_tile_size * j) + (tx * d) + x] = dKj[(tx * d) + x];
            dV[qkv_offset + (row_tile_size * j) + (tx * d) + x] = dVj[(tx * d) + x];
        }
    }
}

std::vector<torch::Tensor> flash_attention_2_forward(torch::Tensor Q, torch::Tensor K, torch::Tensor V) {
    // TODO: determine Bc, Br dynamically
    const int Bc = 32; const int Br = 32;

    const int B = Q.size(0); const int nh = Q.size(1);
    const int N = Q.size(2); const int d = Q.size(3);

    const int Tc = ceil((float) N / Bc); const int Tr = ceil((float) N / Br);
    const float softmax_scale = 1.0 / sqrt(d);

    // Initialize O, L to HBM
    auto O = torch::zeros_like(Q);
    auto L = torch::zeros({B, nh, N});
    torch::Device device(torch::kCUDA);
    L = L.to(device);

    // Calculate SRAM size needed per block
    int col_tile_size = Bc * d;  // size of Kj, Vj
    int row_tile_size = Br * d;  // size of Qi
    const int sram_size =
        (2 * col_tile_size * sizeof(float))  // SRAM size for Kj, Vj
        + (row_tile_size * sizeof(float))  // SRAM size for Qi
        + (Bc * Br * sizeof(float));  // SRAM size for S
    int max_sram_size;
    hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    printf("Max shared memory: %d, requested shared memory: %d \n", max_sram_size, sram_size);

    dim3 grid_dim(B, nh);  // batch_size x num_heads
    dim3 block_dim(Br);  // Br threads per block

    flash_attention_2_forward_kernel<<<grid_dim, block_dim, sram_size>>>(
        Q.data_ptr<float>(), K.data_ptr<float>(), V.data_ptr<float>(),
        N, d, Tc, Tr, Bc, Br, softmax_scale,
        L.data_ptr<float>(), O.data_ptr<float>()
    );
    return {O, L};
}

std::vector<torch::Tensor> flash_attention_2_backward(
    torch::Tensor Q,
    torch::Tensor K,
    torch::Tensor V,
    torch::Tensor O,
    torch::Tensor dO,
    torch::Tensor L
) {
    // TODO: determine Bc, Br dynamically
    const int Bc = 16; const int Br = 16;

    const int B = Q.size(0); const int nh = Q.size(1);
    const int N = Q.size(2); const int d = Q.size(3);

    const int Tc = ceil((float) N / Bc); const int Tr = ceil((float) N / Br);
    const float softmax_scale = 1.0 / sqrt(d);

    auto dQ = torch::zeros_like(Q);
    auto dK = torch::zeros_like(K);
    auto dV = torch::zeros_like(V);

    // Calculate SRAM size needed per block
    int col_tile_size = Bc * d;  // size of Kj, Vj
    int row_tile_size = Br * d;  // size of Qi, Oi, dOi
    const int sram_size =
        (4 * col_tile_size * sizeof(float))  // SRAM size for Kj, Vj, dKj, dVj
        + (3 * row_tile_size * sizeof(float))  // SRAM size for Qi, Oi, dOi
        + (2 * Br * Bc * sizeof(float));  // SRAM size for S, dS
    int max_sram_size;
    hipDeviceGetAttribute(&max_sram_size, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    printf("Max shared memory: %d, requested shared memory: %d \n", max_sram_size, sram_size);

    dim3 grid_dim(B, nh);  // batch_size x num_heads
    dim3 block_dim(Br);  // Bc threads per block

    flash_attention_2_backward_kernel<<<grid_dim, block_dim, sram_size>>>(
        Q.data_ptr<float>(), K.data_ptr<float>(), V.data_ptr<float>(),
        O.data_ptr<float>(), dO.data_ptr<float>(),
        L.data_ptr<float>(),
        N, d, Tc, Tr, Bc, Br, softmax_scale,
        dQ.data_ptr<float>(), dK.data_ptr<float>(), dV.data_ptr<float>()
    );
    return {dQ, dK, dV};
}
